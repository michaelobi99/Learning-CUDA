#include "hip/hip_runtime.h"

#include <stdio.h>

void addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    printf("idx i = %d\n", i);
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int* c = (int*)malloc(arraySize * sizeof(int));


    // Add vectors in parallel.
    addWithCuda(c, a, b, arraySize);

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    free(c);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;


    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

}
